#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>
#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <random>
#include <set>
#include <chrono>

#include "util.h"
#include "official.h"

#define PROBLEM_SIZE 48
#define POPULATION_SIZE (PROBLEM_SIZE * PROBLEM_SIZE) * 1 //must be multiple of PROBLEM_SIZE ^ 2
#define EPOCHS 100
#define ISLANDS 4

using namespace std;

#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}

/* Function to evaluate fitness using a population matrix, the node edge incidence matrix and the thread numbers to get the correct members */
void evaluate_fitness(const int *population, float *fitness, float *nei, int xthr, int ythr, float *fitness_sum, float *fitness_max) {
    fitness_sum[0] = 0.0;
    for(int i = 0; i < POPULATION_SIZE/(PROBLEM_SIZE * PROBLEM_SIZE), i++){
        float route_distance = 0.0;
        for(int j = 0; j < PROBLEM_SIZE - 1; j++){
            route_distance += nei[population[j + xthr * (ythr + PROBLEM_SIZE) * PROBLEM_SIZE * i] + PROBLE_SIZE * individual[j + 1 + xthr * (ythr + PROBLEM_SIZE) * PROBLEM_SIZE * i]];   //matrix lookup for a distance between two cities
        }
        route_distance += nei[population[PROBLEM_SIZE - 1 + xthr * (ythr + PROBLEM_SIZE) * PROBLEM_SIZE * i] + PROBLE_SIZE * individual[0 + xthr * (ythr + PROBLEM_SIZE) * PROBLEM_SIZE * i]];
	fitness[xthr * (ythr + PROBLEM_SIZE) * i] = 1/route_distance;
	__syncthreads();
	if (fitness[xthr * (ythr + PROBLEM_SIZE) * i] > fitness_max[0]){
		fitness_max[0] = fitness[xthr * (ythr + PROBLEM_SIZE) * i];
	}
	fitness_sum[0] += route_distance;
    }
}

/* Function that performs crossover based on roulette wheel selection  */
void crossover(int *population, float *fitness, int xthr, int ythr, float *fitness_sum, int *new_population){
   int* temp_population = new int[PROBLEM_SIZE * POPULATION_SIZE];
   auto dist = discrete_distribution<int>(fitness);
   int parent1, parent2;
   volatile int vol_index = 0;
   int index;
   random_device rd;
   gen = mt19937(rd());

   for(int i = 0; i < POPULATION_SIZE/(PROBLEM_SIZE * PROBLEM_SIZE), i++){
	parent1 = dist(gen);
	parent2 = dist(gen);
	while(parent2 == parent1){
		parent2 = dist(gen);
	}
	index = vol_index;
	vol_index = vol_index + 2;;
	for 
	
	
	
   // Breed any random individuals
   for (int i = this->elite_size; i < this->population_count; ++i) {
   int rand1 = dist(gen);
   int rand2 = dist(gen);
   int* parent1 = new int[this->problem_size];
   int* parent2 = new int[this->problem_size];
   while (rand1 == rand2) {
       rand2 = dist(gen);
   }
   parent1 = this->getGene(rand1);
   parent2 = this->getGene(rand2);
       this->breed(
               parent1,
               parent2,
               temp_population[i]);

   }

   for (int i = 0; i < this->population_count; ++i) {
       for (int j = 0; j < this->problem_size; ++j) {
           this->population[i + this->population_count * j] = temp_population[i][j]; //this doesnt work
       // cout << this->population[i + this->population_count * j] << endl;
       }
   }





    int geneA  = rand_range(0, PROBLEM_SIZE - 1);
    int geneB = rand_range(0, PROBLEM_SIZE - 1);
    int startGene = min(geneA, geneB);
    int endGene = max(geneA, geneB);

    set<int> selected;
    for (int i = startGene; i <= endGene; ++i) {
        child[i] = parent1[i];
        selected.insert(parent1[i]);
    }

    int index = 0;
    for (int i = 0; i < PROBLEM_SIZE; ++i) {
        // If not already chosen that city
        if (selected.find(parent2[i]) == selected.end()) {
            if (index == startGene) {
                index = endGene + 1;
            }
            child[index] = parent2[i];
            index++;
        }
    }
}

__global__ void kernel_tsp_ga(float *pInputs, float *pOutputs) {
    int Inx = blockIdx.x; //island
    int Iny = blockIdx.y; //island
    int Inz = blockIdx.z; //island
    int x_thread = threadIdx.y; //gene (column) of population
    int y_thread = threadIdx.x; //member (row) of group
    int z_thread = threadIdx.z; //group of consecutive members (rows) of population
    extern __shared__ float input[];
    extern __shared__ int population[];
    extern __shared__ int new_population[];
    extern __shared__ float fitness[];
    extern __shared__ volatile float fitness_sum = 0;
    extern __shared__ volatile float fitness_max = 0;
    extern __shared__ float best_individual_per_generation[];
 
    random_device rd;
    gen = mt19937(rd());
 
    //copy lookup matrix into shared memory (to be seen by all blocks)
    input[x_thread + y_thread * PROBLEM_SIZE] = pInputs[x_thread + y_thread * PROBLEM_SIZE];
    
    //instantiate population (no syncthreads needed) POPULATION_SIZE must be divisible by PROBLEM_SIZE
    vector<int> tmp_indices(problem_size);
    tmp_indices[x_thread] = x_thread;
    __syncthreads();
    for(int i = 0; i < POPULATION_SIZE/(PROBLEM_SIZE * PROBLEM_SIZE); i++){
        shuffle(tmp_indices.begin(), tmp_indices.end(), gen);
        for (int z = 0; z < PROBLEM_SIZE; z++){
            population[z + x_thread * y_thread * PROBLEM_SIZE * i] = tmp_indices[x_thread];
        }
    }
    __syncthreads();
    
    //start GA
    for (i = 0; i < EPOCHS){
        //fitness evaluation
	fitness_sum = 0;
	fitness_min = 0;
	__syncthreads();
        evaluate_fitness(population, fitness, input, x_thread, y_thread, &fitness_sum, &fitness_max);
        __syncthreads();
	best_fitness_per_generation[i] = fitness_max;
        
        //crossover
        
    }
}

int traveling_salesman_problem() {

	float *input_ = get_parameter(inputName, PROBLEM_SIZE * PROBLEM_SIZE);
	float *output_;
    
	float *input, *population;
	uint64_t nT1 = 0, nT2 = 0;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////


	/*  1. Data preparation  */

	int nInput = PROBLEM_SIZE * PROBLEM_SIZE;
	int nPopulation = PROBLEM_SIZE * POPULATION_SIZE;

	hipMalloc((void **) &input, nInput*sizeof(float));
	hipMalloc((void **) &population, nPopulation*sizeof(float));
	hipMemset((void *) input, 0, nInput * sizeof(float));
	hipMemset((void *) output, 0, nOutput*sizeof(float));
	hipMemcpy(input, input_, nInput*sizeof(float), hipMemcpyHostToDevice); //from input_ to input
	
	float output[nPopulation];

	

	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();
	//hipProfilerStart();

	//###B'dB the first matrix multiplication for data transform###
	kernel_tsp <<<ISLANDS, dim3(PROBLEM_SIZE, PROBLEM_SIZE)>>> (input, output);
	hipDeviceSynchronize();

	hipError_t errorBTDB = hipGetLastError();
	if (errorBTDB != hipSuccess) {
  		fprintf(stderr, "ERROR BTDB: %s \n", hipGetErrorString(errorBTDB));
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_tsp_ga), hipFuncCachePreferShared); //maybe not necessary

	//hipProfilerStop();
	nT2 = getTimeMicroseconds64();

	printf("TotalTime = %lu us\n", nT2-nT1); 


	/*  3. Copy back and free  */
	s = hipMemcpy(output_, output, nOutput*sizeof(float), hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	//cudaCheckError();

	hipFree(output);

	return ((nT2-nT1) << 16);
}
