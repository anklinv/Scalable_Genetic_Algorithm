#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <random>
#include <set>
#include <chrono>
#include <array>
#include <cassert>
#include <math.h>

#include <hiprand/hiprand_kernel.h>

#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>

// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

#define POP_SIZE 1000
#define PROB_SIZE 48
#define EPOCHS 100
#define ISLANDS 4
#define MUTATION_RATE 0.15
#define MIGRATION 5    //integer determining how many members migrate from one island
#define POP(i,j) population[i * PROB_SIZE + j]
#define NPOP(i,j) new_population[i * PROB_SIZE + j]
#define DIST(i,j) problem[i * PROB_SIZE + j]
#define ISLAND_POP_SIZE POP_SIZE / ISLANDS

typedef float problem_t[PROB_SIZE][PROB_SIZE];

typedef int individual_t[PROB_SIZE];
typedef individual_t population_t[POP_SIZE];

typedef float fitness_t;
typedef fitness_t population_fitness_t[POP_SIZE];

//https://stackoverflow.com/questions/1577475/c-sorting-and-keeping-track-of-indexes
//https://github.com/thrust/thrust/wiki/Quick-Start-Guide#fancy-iterators

using namespace std;

__global__ void rank_individuals(
    population_t population,
    problem_t problem,
    population_fitness_t fitness
) {
	//blockDim: number of threads in block
	//gridDim: number of blocks in grid
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    individual_t &i = population[index];
    float distance = 0;
    for (int j = 0; j < PROB_SIZE - 1; j++) {
        distance += problem[i[j]][i[j + 1]];
    }
    distance += problem[i[PROB_SIZE - 1]][i[0]];
    fitness[index] = 1 / distance;
	__syncthreads();
}

__global__ void crossover_and_migrate(
    population_t population,
    population_fitness_t fitness,
    population_fitness_t fitness_prefix_sum,
    population_t new_population
) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    individual_t &new_individual = new_population[index];
    //get indices of the two parents by a random variable that has the normalized fitness values as distributions (use the thrust lower bound on the prefix sorted fitness values)
    hiprandState s;
    hiprand_init(index, 0, 0, &s);
    // TODO search within a specific single island
    int parent_1_idx = thrust::lower_bound(
        thrust::device,
        fitness_prefix_sum,
        // fitness_prefix_sum + ISLAND_POP_SIZE,
        fitness_prefix_sum + POP_SIZE,
        hiprand_uniform(&s) * fitness_prefix_sum[POP_SIZE-1]
    ) - fitness_prefix_sum;
    int parent_2_idx = parent_1_idx; // TODO when parent1 is correct, copy code to parent2
    // int parent2_id = thrust::lower_bound(
    //     thrust::device, fitness_prefix_sum, fitness_prefix_sum + ISLAND_POP_SIZE, hiprand_uniform(curand_state)
    // ) - fitness_prefix_sum - 1;
    individual_t &parent_1 = population[parent_1_idx];
    individual_t &parent_2 = population[parent_2_idx];
	
    //fill the first half of the child with the first parent
    for (int j = 0; j < PROB_SIZE / 2; j++) {
        new_individual[j] = parent_1[j];
    }
		
    //fill the rest of the child with -1 entries such that it is no longer garbage
    // TODO is that even necessary?
    for (int j = PROB_SIZE / 2; j < PROB_SIZE; j++) {
        new_individual[j] = -1;
    }
		
    //for the second half of the child we only choose entries from parent 2 that aren't already in the child
    // TODO implement with masking
    for (int j = PROB_SIZE / 2; j < PROB_SIZE; j++) {
        for (int k = 0; k < PROB_SIZE; k++) {
            int existing = thrust::find(
                thrust::device,
                new_individual,
                new_individual + j,
                parent_2[k]
            ) - new_individual;
            // if this element does not feature in new_individual, add it
            if (existing == j) {
                new_individual[j] = existing;
                break;
            }
        }
    }
    // //perform the migration
    // if (threadIdx.x == 0) {
    //     island_1 = index % ISLANDS;
    //     island_2 = (island_1 + 1) % ISLANDS;
    //     r_1 = rand() % ISLAND_POP_SIZE;
    //     r_2 = rand() % ISLAND_POP_SIZE;
    // }
	// __syncthreads();
    // //Each thread takes a single gene from one of five
    // int temp[PROB_SIZE * MIGRATION];
    // if (threadIdx.x != 0 && threadIdx.x < PROB_SIZE * MIGRATION) {
    //     int migrant_index = threadIdx.x - 1;
    //     // TODO where does island_1, r_1 etc come from? are they shared?
    //     temp[migrant_index] = NPOP(island_1 * ISLAND_POP_SIZE + r_1, remainder(migrant_index, PROB_SIZE));
    //     NPOP(island_1 * POP_SIZE / ISLAND + r_1, remainder(migrant_index, PROB_SIZE)) = NPOP(island_2 * ISLAND_POP_SIZE + r_2, remainder(migrant_index, PROB_SIZE))
    //     NPOP(island_2 * POP_SIZE / ISLAND + r_2, remainder(migrant_index, PROB_SIZE)) = temp[migrant_index];
    // }
}

__global__ void mutate(population_t population){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    individual_t &i = population[index];

    hiprandState s;
    hiprand_init(index, 0, 0, &s);

    for (int j = 0; j < PROB_SIZE; j++){
        for (int k = 0; k < PROB_SIZE; k++) {
            // TODO this will favour early elements
            if (hiprand_uniform(&s) < MUTATION_RATE) {
                int temp = i[j];
                i[j] = i[k];
                i[k] = temp;
                break;
            }
        }
    }
}

int main (void) {

    //arrays needed for problem
    individual_t *population, *new_population; // population_t
    float (*problem)[PROB_SIZE]; // problem_t
	float *fitness, *fitness_prefix_sum; // population_fitness_t
    // int *sorted_fitness;

	//random device
	random_device rd;
	mt19937 gen = mt19937(rd());

	//Unified memory (accessible from CPU and GPU)
	hipMallocManaged(&problem, sizeof(problem_t));
	hipMallocManaged(&population, sizeof(population_t));
	hipMallocManaged(&new_population, sizeof(population_t));
	hipMallocManaged(&fitness, sizeof(population_fitness_t));
	hipMallocManaged(&fitness_prefix_sum, sizeof(population_fitness_t));
	// hipMallocManaged(sorted_fitness, POP_SIZE *sizeof(int));

	//initialize sequence of matrix indices to be used for initializing the population
	vector<int> tmp_indices(PROB_SIZE);
    	for (int i = 0; i < PROB_SIZE; ++i) {
        	tmp_indices[i] = i;
    	}

    //initialize problem (random for now, from file later)
    float problem_x[PROB_SIZE], problem_y[PROB_SIZE];
    uniform_real_distribution<float> dis(0.0f, 100.0f);
	for (int i = 0; i < PROB_SIZE; i++) {
        problem_x[i] = dis(gen);
        problem_y[i] = dis(gen);
    }
    for (int i = 0; i < PROB_SIZE; i++) {
        for (int j = 0; j < PROB_SIZE; j++) {
            problem[i][j] = sqrt(
                (problem_x[j] - problem_x[i]) * (problem_x[j] - problem_x[i]) +
                (problem_y[j] - problem_y[i]) * (problem_y[j] - problem_y[i])
            );
        }
    }

	//initialize population (random) and fitness (zero)
	for (int i = 0; i < POP_SIZE; ++i) {
        	shuffle(tmp_indices.begin(), tmp_indices.end(), gen);
        	for (int j = 0; j < PROB_SIZE; ++j) {
            		population[i][j] = tmp_indices[j];
        	}
		fitness[i] = 0.0f;
		// sorted_fitness[i] = i;
    }

	//run GA
	for (int i = 0; i < EPOCHS; i++){
		//begin epoch by ranking fitness of population
		rank_individuals<<<ISLANDS, ISLAND_POP_SIZE>>>(population, problem, fitness);
		
		//perform a sort of the fitness values to implement the elitism strategy
        // for (int j = 0; j < ISLANDS; j++) {
        //     thrust::sort_by_key(sorted_fitness + j * ISLAND_POP_SIZE, sorted_fitness + (j + 1) * ISLAND_POP_SIZE, fitness + j * ISLAND_POP_SIZE);
        // }

        //perform a prefix sum of the fitness in order to easily perform roulette wheel selection of a suitable parent
        // TODO calculate inclusive scan within islands
        // for (int j = 0; j < ISLANDS; j++) {
        //     thrust::inclusive_scan(fitness + j * ISLAND_POP_SIZE, fitness + (j + 1) * ISLAND_POP_SIZE, fitness_prefix_sum + j * ISLAND_POP_SIZE);
        // }
        thrust::inclusive_scan(fitness, fitness + POP_SIZE, fitness_prefix_sum);
        
		//perform crossover on the islands
		crossover_and_migrate<<<ISLANDS, ISLAND_POP_SIZE>>>(population, fitness, fitness_prefix_sum, new_population);
  
        //mutate the population
        mutate<<<ISLANDS, ISLAND_POP_SIZE>>>(population);

        // float best_fitness = 0.0f;
        cout << fitness[0] << endl;
	}

	hipFree(problem);
	hipFree(population);
	hipFree(new_population);
	hipFree(fitness);
	hipFree(fitness_prefix_sum);
	// hipFree(sorted_fitness);
	
	return 0;
}
