#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <random>
#include <set>
#include <chrono>
#include <array>
#include <cassert>
#include <math.h>

#include <hiprand/hiprand_kernel.h>

#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>

// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

#define POP_SIZE 1000
#define PROB_SIZE 48
#define EPOCHS 100
#define ISLANDS 4
#define MUTATION_RATE 0.15
#define MIGRATION 5    //integer determining how many members migrate from one island
#define POP(i,j) population[i * PROB_SIZE + j]
#define NPOP(i,j) new_population[i * PROB_SIZE + j]
#define DIST(i,j) problem[i * PROB_SIZE + j]
#define ISLAND_POP_SIZE (POP_SIZE / ISLANDS)

typedef float problem_t[PROB_SIZE][PROB_SIZE];

typedef int individual_t[PROB_SIZE];
typedef individual_t population_t[POP_SIZE];

typedef float fitness_t;
typedef fitness_t population_fitness_t[POP_SIZE];

//https://stackoverflow.com/questions/1577475/c-sorting-and-keeping-track-of-indexes
//https://github.com/thrust/thrust/wiki/Quick-Start-Guide#fancy-iterators

using namespace std;

__global__ void rank_individuals(
    population_t population,
    problem_t problem,
    population_fitness_t fitness
) {
	//blockDim: number of threads in block
	//gridDim: number of blocks in grid
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    individual_t &i = population[index];
    float distance = 0;
    for (int j = 0; j < PROB_SIZE - 1; j++) {
        distance += problem[i[j]][i[j + 1]];
    }
    distance += problem[i[PROB_SIZE - 1]][i[0]];
    fitness[index] = 1 / distance;
	__syncthreads();
}

__global__ void crossover_and_migrate(
    population_t population,
    population_fitness_t fitness,
    population_fitness_t fitness_prefix_sum,
    population_t new_population
) {
    int island = blockIdx.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    individual_t &new_individual = new_population[index];

    //get indices of the two parents by a random variable that has the normalized fitness values as distributions (use the thrust lower bound on the prefix sorted fitness values)
    hiprandState s;
    hiprand_init(index, 0, 0, &s);

    int parent_1_idx = thrust::lower_bound(
        thrust::device,
        fitness_prefix_sum + island * ISLAND_POP_SIZE,
        fitness_prefix_sum + (island + 1) * ISLAND_POP_SIZE,
        hiprand_uniform(&s) * fitness_prefix_sum[POP_SIZE-1]
    ) - fitness_prefix_sum;
    individual_t &parent_1 = population[parent_1_idx];

    int parent_2_idx = thrust::lower_bound(
        thrust::device,
        fitness_prefix_sum + island * ISLAND_POP_SIZE,
        fitness_prefix_sum + (island + 1) * ISLAND_POP_SIZE,
        hiprand_uniform(&s) * fitness_prefix_sum[POP_SIZE-1]
    ) - fitness_prefix_sum;
    individual_t &parent_2 = population[parent_2_idx];
	
    //fill the first half of the child with the first parent
    for (int j = 0; j < PROB_SIZE / 2; j++) {
        new_individual[j] = parent_1[j];
    }

    //for the second half of the child we only choose entries from parent 2 that aren't already in the child
    // TODO implement with masking
    for (int j = PROB_SIZE / 2; j < PROB_SIZE; j++) {
        for (int k = 0; k < PROB_SIZE; k++) {
            int existing = thrust::find(
                thrust::device,
                new_individual,
                new_individual + j,
                parent_2[k]
            ) - new_individual;
            // if this element does not already feature in new_individual, add it
            if (existing == j) {
                new_individual[j] = parent_2[k];
                break;
            }
        }
    }

    // //perform the migration
    // int target_island = (island + 1) % ISLANDS;
    // __shared__ int r_1, r_2;
    // if (threadIdx.x == 0) {
    //     r_1 = rand() % ISLAND_POP_SIZE;
    //     r_2 = rand() % ISLAND_POP_SIZE;
    // }
	__syncthreads();
    // //Each thread takes a single gene from one of five
    // int temp[PROB_SIZE * MIGRATION];
    // if (threadIdx.x != 0 && threadIdx.x < PROB_SIZE * MIGRATION) {
    //     int migrant_index = threadIdx.x - 1;
    //     temp[migrant_index] = NPOP(island_1 * ISLAND_POP_SIZE + r_1, remainder(migrant_index, PROB_SIZE));
    //     NPOP(island_1 * POP_SIZE / ISLAND + r_1, remainder(migrant_index, PROB_SIZE)) = NPOP(island_2 * ISLAND_POP_SIZE + r_2, remainder(migrant_index, PROB_SIZE))
    //     NPOP(island_2 * POP_SIZE / ISLAND + r_2, remainder(migrant_index, PROB_SIZE)) = temp[migrant_index];
    // }

    // overwrite existing population with the newly created one
    for (int j = 0; j < PROB_SIZE; j++) {
        population[index][j] = new_individual[j];
    }
}

__global__ void mutate(population_t population){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    individual_t &i = population[index];

    hiprandState s;
    hiprand_init(index, 0, 0, &s);

    for (int j = 0; j < PROB_SIZE; j++){
        for (int k = 0; k < PROB_SIZE; k++) {
            // TODO this will favour early elements
            if (hiprand_uniform(&s) < MUTATION_RATE) {
                int temp = i[j];
                i[j] = i[k];
                i[k] = temp;
                break;
            }
        }
    }
}

int main (void) {

    //arrays needed for problem
    individual_t *population, *new_population; // population_t
    float (*problem)[PROB_SIZE]; // problem_t
	float *fitness, *fitness_prefix_sum; // population_fitness_t
    int *sorted_fitness_idxs;

	//random device
	random_device rd;
	mt19937 gen = mt19937(rd());

	//Unified memory (accessible from CPU and GPU)
	hipMallocManaged(&problem, sizeof(problem_t));
	hipMallocManaged(&population, sizeof(population_t));
	hipMallocManaged(&new_population, sizeof(population_t));
	hipMallocManaged(&fitness, sizeof(population_fitness_t));
	hipMallocManaged(&fitness_prefix_sum, sizeof(population_fitness_t));
	hipMallocManaged(&sorted_fitness_idxs, POP_SIZE * sizeof(int));

	//initialize sequence of matrix indices to be used for initializing the population
	vector<int> tmp_indices(PROB_SIZE);
    	for (int i = 0; i < PROB_SIZE; ++i) {
        	tmp_indices[i] = i;
    	}

    //initialize problem (random for now, from file later)
    float problem_x[PROB_SIZE], problem_y[PROB_SIZE];
    uniform_real_distribution<float> dis(0.0f, 100.0f);
	for (int i = 0; i < PROB_SIZE; i++) {
        problem_x[i] = dis(gen);
        problem_y[i] = dis(gen);
    }
    for (int i = 0; i < PROB_SIZE; i++) {
        for (int j = 0; j < PROB_SIZE; j++) {
            problem[i][j] = sqrt(
                (problem_x[j] - problem_x[i]) * (problem_x[j] - problem_x[i]) +
                (problem_y[j] - problem_y[i]) * (problem_y[j] - problem_y[i])
            );
        }
    }

	//initialize population (random) and fitness (zero)
	for (int i = 0; i < POP_SIZE; ++i) {
        	shuffle(tmp_indices.begin(), tmp_indices.end(), gen);
        	for (int j = 0; j < PROB_SIZE; ++j) {
            		population[i][j] = tmp_indices[j];
        	}
		fitness[i] = 0.0f;
    }

	//run GA
	for (int epoch = 0; epoch < EPOCHS; epoch++){
		//begin epoch by ranking fitness of population
		rank_individuals<<<ISLANDS, ISLAND_POP_SIZE>>>(population, problem, fitness);
        
        // TODO this actually changes order of fitness, but not population
		// //perform a sort of the fitness values to implement the elitism strategy
        // for (int i = 0; i < POP_SIZE; ++i) {
        //     sorted_fitness_idxs[i] = i;
        // }
        // for (int j = 0; j < ISLANDS; j++) {
        //     thrust::sort_by_key(
        //         thrust::device,
        //         fitness + j * ISLAND_POP_SIZE,
        //         fitness + (j + 1) * ISLAND_POP_SIZE,
        //         sorted_fitness_idxs + j * ISLAND_POP_SIZE
        //     );
        // }

        //perform a prefix sum of the fitness in order to easily perform roulette wheel selection of a suitable parent
        for (int j = 0; j < ISLANDS; j++) {
            thrust::inclusive_scan(
                thrust::device,
                fitness + j * ISLAND_POP_SIZE,
                fitness + (j + 1) * ISLAND_POP_SIZE,
                fitness_prefix_sum + j * ISLAND_POP_SIZE
            );
        }
        
		//perform crossover on the islands
		crossover_and_migrate<<<ISLANDS, ISLAND_POP_SIZE>>>(population, fitness, fitness_prefix_sum, new_population);
  
        //mutate the population
        mutate<<<ISLANDS, ISLAND_POP_SIZE>>>(population);

        // float best_fitness = 0.0f;
        cout << "epoch " << epoch << ": " << fitness[0] << endl;
	}

	hipFree(problem);
	hipFree(population);
	hipFree(new_population);
	hipFree(fitness);
	hipFree(fitness_prefix_sum);
	hipFree(sorted_fitness_idxs);
	
	return 0;
}
